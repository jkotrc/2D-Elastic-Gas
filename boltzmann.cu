#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define mag(x,y) sqrt(x*x+y*y)
#define mag2(x,y) x*x+y*y

#define BLOCK_SIZE 512


__global__ void step_kernel(float *posx, float *posy, float *vx, float *vy, int N, float size, float epsilon, float width, float height) {
			int idx = blockIdx.x*blockDim.x+threadIdx.x;
			//printf("%u, ",idx);

			const float sqrtarg=1+8*idx;
			int j = (int)(sqrt(sqrtarg)+1);
			j/=2;
			int k = idx - (j*(j-1))/2;

			if (j >= N || k >= N){return;}

			float magnitude=mag((posx[k]-posx[j]),(posy[k]-posy[j]));
			if (magnitude <= size) {
					float dot = ((vx[k]-vx[j])*(posx[k]-posx[j]))+((vy[k]-vy[j])*(posy[k]-posy[j]));
					float mg = size*size;
					vx[k] -= (posx[k]-posx[j]) * (dot/mg);
					vy[k] -= (posy[k]-posy[j]) * (dot/mg);
					vx[j] -= (posx[j]-posx[k]) * (dot/mg);
					vy[j] -= (posy[j]-posy[k]) * (dot/mg);

					//when balls get stuck, push one of them just outside the other
					posx[j] -= (size/magnitude-0.8)*(posx[k]-posx[j]);
					posy[j] -= (size/magnitude-0.8)*(posy[k]-posy[j]);
				}

			if (posx[j] >= width-size/2) {
				vx[j] *= -1;
				posx[j] = width-size/2;
			}
			if (posx[j] <= -width+size/2) {
				vx[j] *= -1;
				posx[j] = -width+size/2;
			}
			if (posy[j] >= height-size/2) {
				vy[j] *= -1;
				posy[j] = height-size/2;
			}
			if (posy[j] <= -height+size/2) {
				vy[j] *= -1;
				posy[j] = -height+size/2;
			}
			
			if (posx[k] >= width-size/2) {
				vx[k] *= -1;
				posx[k] = width-size/2;
			}
			if (posx[k] <= -width+size/2) {
				vx[k] *= -1;
				posx[k] = -width+size/2;
			}
			if (posy[k] >= height-size/2) {
				vy[k] *= -1;
				posy[k] = height-size/2;
			}
			if (posy[k] <= -height+size/2) {
				vy[k] *= -1;
				posy[k] = -height+size/2;
			}

			posx[j]+=epsilon*vx[j];
			posy[j]+=epsilon*vy[j];

			posx[k]+=epsilon*vx[k];
			posy[k]+=epsilon*vy[k];
}